#include "hip/hip_runtime.h"
#include "softmaxloss.h"

__global__ void crossbowKernelSoftMaxLossCompute (
	const int nthreads,
	const float* prob_data,
	const int* label,
	float* loss,
	const int num,
	const int dim,
	const int spatial_dim,
	const bool has_ignore_label_,
	const int ignore_label_,
	float* counts) {

	CUDA_KERNEL_LOOP(index, nthreads) {
		const int n = index / spatial_dim;
		const int s = index % spatial_dim;
		const int label_value = label[n * spatial_dim + s];
		if (has_ignore_label_ && label_value == ignore_label_) {
			loss[index] = 0;
			counts[index] = 0;
		} else {
			loss[index] = -log(max(prob_data[n * dim + label_value * spatial_dim + s], FLT_MIN));
			counts[index] = 1;
		}
	}
}

void crossbowKernelSoftMaxLoss (void *args) {

	/* Kernel configuration parameters */
	int ignorelabelvalue;
	bool ignorelabel;

	int outer, inner, dim, nthreads;

	float alpha;
	float *C;

	/* Input and output variables */
	crossbowVariableP theInput, theLabels;
	crossbowDataBufferP input, labels;
	crossbowDataBufferP output;
	int labels_offset;

	/* Local variables */
	crossbowDataBufferP losses, counts;
	int losses_length, counts_length;
	
	/* struct hipPointerAttribute_t attributes; */
	
	crossbowStreamP s = (crossbowStreamP) args;

	/* checkCublasStatus(hipblasSetStream (s->cublasHandle, s->stream)); */

	/* Get input variable */
	theInput = (crossbowVariableP) s->op->kernel->inputs[0];

	if (crossbowDataflowMostUpstream(s->dataflow, s->op))
		illegalStateException();

	input = crossbowStreamGetCurrentInput (s);

	/* Get labels */
	theLabels = (crossbowVariableP) s->op->kernel->inputs[1];
	labels = crossbowVariableGetDataBuffer (s->labels, &labels_offset, NULL);
	
	/*
	checkCudaErrors(hipPointerGetAttributes (&attributes, labels->dev));
	info("labels->dev at %p: device %d device pointer %p host pointer %p managed %d\n", 
        labels->dev, attributes.device, attributes.devicePointer, attributes.hostPointer, attributes.isManaged);
	*/
	
	/* Get read-write local variables */
	losses = crossbowLocalVariableGetDataBuffer ((crossbowLocalVariableP) crossbowArrayListGet (s->op->kernel->variables, 0), s->deviceId, s->id, NULL, &losses_length);
	counts = crossbowLocalVariableGetDataBuffer ((crossbowLocalVariableP) crossbowArrayListGet (s->op->kernel->variables, 1), s->deviceId, s->id, NULL, &counts_length);

	/* Get an output buffer */
	output = crossbowStreamGetCurrentOutput (s);

	/* Get kernel configuration parameters */
	ignorelabelvalue = crossbowKernelConfigParamGetIntValue ((crossbowKernelConfigParamP) crossbowArrayListGet(s->op->kernel->parameters, 0));
	ignorelabel = (ignorelabelvalue >= 0);

	nthreads = theLabels->schema->elements;
	outer = crossbowVariableSchemaCountElementsInRange (theInput->schema, 0, 1);
	inner = crossbowVariableSchemaCountElementsFrom (theInput->schema, 2);
	dim = theInput->schema->elements / outer;

#ifndef KERNEL_NOOP
	crossbowKernelSoftMaxLossCompute<<<GET_BLOCKS(nthreads), CUDA_NUM_THREADS, 0, s->stream[s->op->branch]>>>(
        nthreads, 
        (float *) (input->dev), 
        (int *) (labels->dev) + (labels_offset / 4), 
        (float *) (losses->dev), 
        outer, 
        dim, 
        inner, 
        ignorelabel, 
        ignorelabelvalue, 
        (float *) (counts->dev));
#else
    /* Subterfuge unused parameter warnings */
	UNUSED (nthreads);
    UNUSED (input);
    UNUSED (labels);
    UNUSED (labels_offset);
    UNUSED (losses);
    UNUSED (outer);
    UNUSED (dim);
    UNUSED (inner);
    UNUSED (ignorelabel);
    UNUSED (ignorelabelvalue);
    UNUSED (counts);
#endif

	alpha = 1 / (float) nthreads;
#ifndef CUBLAS_NOOP
	checkCublasStatus(hipblasSscal (s->cublasHandle[s->op->branch], nthreads, &alpha, (float *) losses->dev, 1));
#else
    /* Subterfuge unused parameter warnings */
    UNUSED (nthreads);
    UNUSED (alpha);
    UNUSED (losses);
#endif
    
	C = (float *) output->dev;
	
	/*
	checkCudaErrors(hipPointerGetAttributes (&attributes, output->dev));
	
    dbg("output->dev at %p: device %d device pointer %p host pointer %p managed %d\n", 
        output->dev, attributes.device, attributes.devicePointer, attributes.hostPointer, attributes.isManaged);
	
    checkCudaErrors(hipPointerGetAttributes (&attributes, losses->dev));
	
    dbg("losses->dev at %p: device %d device pointer %p host pointer %p managed %d\n", 
        losses->dev, attributes.device, attributes.devicePointer, attributes.hostPointer, attributes.isManaged);
	*/

#ifndef CUBLAS_NOOP
	checkCublasStatus(hipblasSetPointerMode(s->cublasHandle[s->op->branch], HIPBLAS_POINTER_MODE_DEVICE));
	checkCublasStatus(hipblasSasum (s->cublasHandle[s->op->branch], nthreads, (float *) losses->dev, 1, C));
	checkCublasStatus(hipblasSetPointerMode(s->cublasHandle[s->op->branch], HIPBLAS_POINTER_MODE_HOST));
#else
    /* Subterfuge unused parameter warnings */
    UNUSED (nthreads);
    UNUSED (losses);
    UNUSED (C);
#endif
    
	/* Store output in stream */
	crossbowListAppend (s->outputs[s->op->id], output);

	/* Return read-write local variables to kernel when the dataflow execution completes */
	crossbowListAppend (s->locals[s->op->id], counts);
	crossbowListAppend (s->locals[s->op->id], losses);

	return;
}
